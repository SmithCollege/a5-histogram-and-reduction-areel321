
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sys/time.h>

#define SIZE 2000

double get_clock() {
        struct timeval tv; int ok;
        ok = gettimeofday(&tv, (void *) 0);
        if (ok<0){
                printf("gettimeofday error\n");
        }
        return (tv.tv_sec*1.0+tv.tv_usec*1.0E-6);
}

__global__ void sum(int* input, int* out) {

	__shared__ float partialSum[2*SIZE];
	unsigned int t = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;
	partialSum[t] = input[start + t];
	partialSum[blockDim.x+t] = input[start+blockDim.x+t];
	
	for (unsigned int stride = 1;
	stride <= blockDim.x; stride *= 2) {
		__syncthreads();
		if (t%stride == 0) {
			partialSum[2*t] += partialSum[2*t+stride];
			out[0] += partialSum[2*t];
		}
	}



}


int main(){


	double *times = (double*)malloc(sizeof(double) * SIZE);
	int *input = (int*)malloc(sizeof(int)*SIZE);
	int *out = (int *)malloc(sizeof(int) *SIZE);
	int *d_input, *d_out;
	
	hipMallocManaged(&d_out, sizeof(int) * SIZE);
	hipMallocManaged(&d_input, sizeof(int) * SIZE);
	

	for (int i=0;i<SIZE;i++) {
		//input[i] = rand() % 16;
		input[i] = 1;
		//printf("%d ", input[i]);
	}
	//printf("\n");


	//calibrate the clock
	double t0 = get_clock();
	for (int i=0; i<SIZE; i++){
		 times[i] = get_clock();
    }
	double t1 = get_clock();
	printf("time per call: %f nx\n", (1000000000.0 * 
	(t1-t0)/SIZE));

	hipMemcpy(d_input, input, SIZE*sizeof(int), hipMemcpyHostToDevice);

	dim3 block(32);
	dim3 grid((SIZE + block.x - 1)/block.x);
	double start = get_clock();
	sum<<<grid, block>>>(d_input, d_out);
	double end = get_clock();

	hipDeviceSynchronize();

	printf("%s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(out, d_out, sizeof(int) * SIZE, 
	hipMemcpyDeviceToHost);

	for (int i=0;i<SIZE;i++){
	//	printf("%d ", out[i]);
	}

	printf("%d\n", out[0]);

	printf("total time: %f\n", end-start);
	

	free(input);
	free(out);
	free(times);
	hipFree(d_out);
	hipFree(d_input);
	return 0;
}
