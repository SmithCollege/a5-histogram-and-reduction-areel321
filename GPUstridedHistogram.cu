
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 50

double get_clock() {
        struct timeval tv; int ok;
        ok = gettimeofday(&tv, (void *) 0);
        if (ok<0){
                printf("gettimeofday error");
        }
        return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


__global__ void histo_kernel(unsigned char* array, long size, unsigned int* histo, int buckets)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x);

    if (i>=size){
		return;
		}
    unsigned char value = array[i];

    int bin = (value % buckets);
    printf("%d", bin);

    atomicAdd(&histo[bin],1);
    
}

int main(){
int size = 8;
	double *times = (double *)malloc(sizeof(double)*size);
         //calibrate clock
        double t0 = get_clock();
        for (int i=0; i<size; i++){
            times[i] = get_clock();
        }
        double t1 = get_clock();
        printf("time per call: %f nx\n", 
        (1000000000.0 * (t1-t0)/size));
	

	unsigned char* array = (unsigned char*)malloc(sizeof(char)*size);
	unsigned int* histo = 
	(unsigned int*)malloc(sizeof(int)*size);

	for (int i=0;i<size;i++){
		array[i]= i;
		//printf("%d ", array[i]);
	}
	printf("\n");

	unsigned char* dArray;
    hipMalloc(&dArray,size);
    hipMemcpy(dArray,array,size,
    hipMemcpyHostToDevice);

    unsigned int* dHist;
    hipMalloc(&dHist,size * sizeof(int));
    hipMemset(dHist,0,size * sizeof(int));

    dim3 block(32);
    dim3 grid((size + block.x - 1)/block.x);

	hipDeviceSynchronize();
    histo_kernel<<<grid,block>>>(dArray,size,dHist, size);
    hipDeviceSynchronize();

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    hipMemcpy(histo,dHist,size * sizeof(int),
    hipMemcpyDeviceToHost);

    for (int i=0;i<size;i++){
    	printf("%d ", histo[i]);
    }
    printf("\n");

    hipFree(dArray);
    hipFree(dHist);






	//print clock times
	//printf("start: %f, end: %f\n", start_time, end_time);

	free(times);
	return 0;
}
