
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 50
#define size 8
#define buckets 8




__global__ void histo_kernel(unsigned char* array, 
unsigned int* histo)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x);

    if (i>=size){
		return;
		}
    unsigned char value = array[i];

    int bin = (value % buckets);
    printf("%d", bin);

    atomicAdd(&histo[bin],1);
    
}

int main(){
	

	unsigned char* array = (unsigned char*)malloc(sizeof(char)*size);
	unsigned int* histo = 
	(unsigned int*)malloc(sizeof(int)*size);

	for (int i=0;i<size;i++){
		array[i]= i;
		//printf("%d ", array[i]);
	}
	printf("\n");

	unsigned char* dArray;
    hipMalloc(&dArray,size);
    hipMemcpy(dArray,array,size,
    hipMemcpyHostToDevice);

    unsigned int* dHist;
    hipMalloc(&dHist,size * sizeof(int));
    hipMemset(dHist,0,size * sizeof(int));

    dim3 block(32);
    dim3 grid((size + block.x - 1)/block.x);

	hipDeviceSynchronize();
    histo_kernel<<<grid,block>>>(dArray,dHist);
    hipDeviceSynchronize();

    printf("%s\n", hipGetErrorString(hipGetLastError()));

    hipMemcpy(histo,dHist,size * sizeof(int),
    hipMemcpyDeviceToHost);

    for (int i=0;i<size;i++){
    	printf("%d ", histo[i]);
    }
    printf("\n");

    hipFree(dArray);
    hipFree(dHist);



	return 0;
}
