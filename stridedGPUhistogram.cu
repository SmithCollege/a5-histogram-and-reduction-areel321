#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 50

double get_clock() {
        struct timeval tv; int ok;
        ok = gettimeofday(&tv, (void *) 0);
        if (ok<0){
                printf("gettimeofday error");
        }
        return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


int main(){
double *times = (double *)malloc(sizeof(double)*width);
	 //calibrate clock
        double t0 = get_clock();
        for (int i=0; i<width; i++){
            times[i] = get_clock();
        }
        double t1 = get_clock();
        printf("time per call: %f nx\n", (1000000000.0 * (t1-t0\
)/width));

//print clock times
        printf("start: %f, end: %f\n", start_time, end_time);

free(times);
return 0;
}