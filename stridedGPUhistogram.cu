
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>

#define BLOCK_SIZE 128
#define size 10
#define buckets 2

__global__ void hist(unsigned char*array, unsigned int*histo){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	if (i>=stride){
		return;
	}
	while (i<size){
		int value = array[i];
		int bin = value % buckets;
		atomicAdd(&(histo[bin]), 1);
		i+=stride;
	}
}

int main(){
	unsigned char* array = (unsigned char*)malloc(sizeof(char)*size);
	unsigned int* histo = (unsigned int*)malloc(sizeof(int)*size);


	for (int i=0;i<size;i++){
		array[i]=i;
	}
	printf("\n");
	unsigned char*d_array;
	hipMalloc(&d_array, size);
	hipMemcpy(d_array, array, size, hipMemcpyHostToDevice);

	unsigned int*d_histo;
	hipMalloc(&d_histo, buckets*sizeof(int));
	hipMemset(d_histo,0,buckets*sizeof(int));

	dim3 block(32);
	dim3 grid((size+block.x-1)/block.x);

	hipDeviceSynchronize();
	hist<<<grid, block>>>(d_array, d_histo);
	hipDeviceSynchronize();
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(histo, d_histo, buckets*sizeof(int), hipMemcpyDeviceToHost);

	for (int i=0;i<buckets;i++) {
		printf("%d ", histo[i]);
	}
	printf("\n");

	free(array);
	free(histo);
	hipFree(d_array);
	hipFree(d_histo);
	
	
	
return 0;
}
